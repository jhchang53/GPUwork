/*
	test driver

*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)

#define THREADS_PER_BLOCK 3
#define BLOCK_COUNT 4

__global__ void setup_kernel(hiprandStateMRG32k3a *state)
{
  int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  unsigned long long seed = id+293;
  unsigned long long sequence = 1;
  unsigned long long offset = 71;
  hiprand_init(seed,sequence,offset,&state[id]);
};

__global__ void generate_kernel(hiprandStateMRG32k3a *state)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  // unsigned int x;
  double x;
  /* Copy state to local memory for efficiency */
  hiprandStateMRG32k3a localState = state[id];
  /* Generate pseudo-random unsigned ints */
  for(int i = 0; i < 4; i++) {
    // x = curand(&localState);
    // printf(" id=%d x=%d\n",id,x);
    x = hiprand_uniform_double(&localState);
    printf(" id=%d x=%.5lf\n",id,x);
  }
  /* Copy state back to global memory */
  state[id] = localState;
};


int main()
{
  const unsigned int threadsPerBlock = THREADS_PER_BLOCK;
  const unsigned int blockCount = BLOCK_COUNT;
  const unsigned int totalThreads = threadsPerBlock * blockCount;
  /*  initialize */
  hiprandStateMRG32k3a *devMRGStates;
  CUDA_CALL(hipMalloc((void **)&devMRGStates, totalThreads *
	sizeof(hiprandStateMRG32k3a)));
  setup_kernel<<<BLOCK_COUNT,THREADS_PER_BLOCK>>>(devMRGStates);
  for(int iter=0; iter < 2; iter++) {
  generate_kernel<<<BLOCK_COUNT,THREADS_PER_BLOCK>>>(devMRGStates);
  hipDeviceSynchronize();
    printf(" iter=%d done\n",iter);
  }
};

